#include "hip/hip_runtime.h"
#include "lib/macros.cuh"
#include "kernel.h"

#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

#define TILE_WIDTH_C BLOCKDIM_X*2
#define TILE_WIDTH_R BLOCKDIM_Y*2

__global__ void cnn_gpu(float* input,
    float* weight,
    float* bias, float* output)
{
  // Input size: 256 x 224 x 224
  // OUtput size: 256 x 112 x 112
  // GPU specs: 16 SMs, 32 blocks/SM, 2048 threads/SM (32K threads total)

  const int nc = kNum / gridDim.z;
  const int channel = blockIdx.z * nc;

  const int br = blockIdx.y * TILE_WIDTH_R;
  const int bc = blockIdx.x * TILE_WIDTH_C;

  const int tr = threadIdx.y;
  const int tc = threadIdx.x;
  const int h = 2 * tr;
  const int w = 2 * tc;

  __shared__ float inputShared [TILE_WIDTH_R+kKernel-1][TILE_WIDTH_C+kKernel-1] __attribute__((aligned(16 * sizeof(float))));
  __shared__ float weightShared[kKernel               ][kKernel               ] __attribute__((aligned(16 * sizeof(float))));

  for (int i = channel; i < channel+nc && i < kNum; i++) {
    // Bias
    float C0 = bias[i];
    float C1 = bias[i];
    float C2 = bias[i];
    float C3 = bias[i];

    // Convolution
    for (int j = 0; j < kNum; j++) {
      // Load input and weight submatrices into shared memory
      inputShared[h  ][w  ] = input(j, br+h  , bc+w  );
      inputShared[h  ][w+1] = input(j, br+h  , bc+w+1);
      inputShared[h+1][w  ] = input(j, br+h+1, bc+w  );
      inputShared[h+1][w+1] = input(j, br+h+1, bc+w+1);

      if (tr < kKernel-1) {
        inputShared[TILE_WIDTH_R+tr][w  ] = input(j, br+TILE_WIDTH_R+tr, bc+w  );
        inputShared[TILE_WIDTH_R+tr][w+1] = input(j, br+TILE_WIDTH_R+tr, bc+w+1);
      }
      if (tc < kKernel-1) {
        inputShared[h  ][TILE_WIDTH_C+tc] = input(j, br+h  , bc+TILE_WIDTH_C+tc);
        inputShared[h+1][TILE_WIDTH_C+tc] = input(j, br+h+1, bc+TILE_WIDTH_C+tc);
      }
      if (tr < kKernel-1 && tc < kKernel-1) {
        inputShared[TILE_WIDTH_R+tr][TILE_WIDTH_C+tc] = input(j, br+TILE_WIDTH_R+tr, bc+TILE_WIDTH_C+tc);
      }
      if (tr < kKernel && tc < kKernel) {
        weightShared[tr][tc] = weight(i, j, tr, tc);
      }
      __syncthreads();

      // Matrix multiplication between weight and input submatrices
      for (int p = 0; p < kKernel; p++) {
        for (int q = 0; q < kKernel; q++) {
          C0 += weightShared[p][q] * inputShared[h+p  ][w+q  ];
          C1 += weightShared[p][q] * inputShared[h+p+1][w+q  ];
          C2 += weightShared[p][q] * inputShared[h+p  ][w+q+1];
          C3 += weightShared[p][q] * inputShared[h+p+1][w+q+1];
        }
      }
      __syncthreads();
    }

    // Max pooling + ReLU
    output(i, (br+h)/2, (bc+w)/2) = max(0.f, max(
        max(C0, C1),
        max(C2, C3)));
  }
}
