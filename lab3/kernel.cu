#include "hip/hip_runtime.h"
#include "lib/macros.cuh"
#include "kernel.h"

#define TILE_WIDTH 32
#define BLOCK_J 8

__global__ void cnn_gpu(float* input,
    float* weight,
    float* bias, float* output)
{
  // Input size: 256 x 224 x 224
  // OUtput size: 256 x 112 x 112
  // GPU specs: 16 SMs, 32 blocks/SM, 2048 threads/SM (32K threads total)

  const int N_CHANNELS = kNum / gridDim.x;

  int bi = blockIdx.x * N_CHANNELS;
  const int tr = threadIdx.y;
  const int tc = threadIdx.x;

  __shared__ float C[TILE_WIDTH][TILE_WIDTH];
  __shared__ float weightShared[BLOCK_J][kKernel][kKernel];
  __shared__ float inputShared[BLOCK_J][TILE_WIDTH+kKernel-1][TILE_WIDTH+kKernel-1];
  
  for (int i = bi; i < bi+N_CHANNELS; i++) {
    for (int h = tr; h < kImSize; h += TILE_WIDTH) {
      for (int w = tc; w < kImSize; w += TILE_WIDTH) {
        // Bias
        float reg = bias[i];

        // Convolution
        for (int bj = 0; bj < kNum; bj += BLOCK_J) {
          for (int j = 0; j < BLOCK_J; j++) {
            inputShared[j][tr][tc] = input(bj+j, h, w);
            if (tc < kKernel-1) {
              inputShared[j][tr][tc+TILE_WIDTH] = input(bj+j, h, w+TILE_WIDTH);
            }
            if (tr < kKernel-1) {
              inputShared[j][tr+TILE_WIDTH][tc] = input(bj+j, h+TILE_WIDTH, w);
            }
            if (tr < kKernel-1 && tc < kKernel-1) {
              inputShared[j][tr+TILE_WIDTH][tc+TILE_WIDTH] = input(bj+j, h+TILE_WIDTH, w+TILE_WIDTH);
            }
            if (tr < kKernel && tc < kKernel) {
              weightShared[j][tr][tc] = weight(i, bj+j, tr, tc);
            }
          }
          __syncthreads();

          for (int j = 0; j < BLOCK_J; j++) {
            for (int p = 0; p < kKernel; ++p) {
              for (int q = 0; q < kKernel; ++q) {
                reg += weightShared[j][p][q] * inputShared[j][tr + p][tc + q];
              }
            }
          }
          __syncthreads();
        }

        // ReLU
        C[tr][tc] = max(0.f, reg);

        // Max pooling
        __syncthreads();
        if (tr % 2 == 0 && tc % 2 == 0) {
          output(i, h/2, w/2) = max(
              max(C[tr][tc    ], C[tr + 1][tc    ]),
              max(C[tr][tc + 1], C[tr + 1][tc + 1]));
        }
      }
    }
  }
}
