#include "hip/hip_runtime.h"
#include "lib/macros.cuh"
#include "kernel.h"

#define TILE_WIDTH 32
#define STEP_J 8

__global__ void cnn_gpu(float* input,
    float* weight,
    float* bias, float* output)
{
  // Input size: 256 x 224 x 224
  // OUtput size: 256 x 112 x 112
  // GPU specs: 16 SMs, 32 blocks/SM, 2048 threads/SM (32K threads total)

  const int NROWS = kNum / gridDim.x;
  const int ROW = blockIdx.x * NROWS;
  const int tr = threadIdx.y;
  const int tc = threadIdx.x;

  __shared__ float C[TILE_WIDTH][TILE_WIDTH];
  __shared__ float weightShared[STEP_J][kKernel][kKernel];
  __shared__ float inputShared[STEP_J][TILE_WIDTH+kKernel-1][TILE_WIDTH+kKernel-1];
  
  for (int i = ROW; i < ROW+NROWS; i++) {
    for (int h = tr; h < kImSize; h += TILE_WIDTH) {
      for (int w = tc; w < kImSize; w += TILE_WIDTH) {
        // Bias
        float reg = bias[i];

        // Convolution
        for (int j = 0; j < kNum; j += STEP_J) {
          for (int jj = 0; jj < STEP_J; jj++) {
            inputShared[jj][tr][tc] = input(j+jj, h, w);
            if (tc < kKernel-1) {
              inputShared[jj][tr][tc+TILE_WIDTH] = input(j+jj, h, w+TILE_WIDTH);
            }
            if (tr < kKernel-1) {
              inputShared[jj][tr+TILE_WIDTH][tc] = input(j+jj, h+TILE_WIDTH, w);
            }
            if (tr < kKernel-1 && tc < kKernel-1) {
              inputShared[jj][tr+TILE_WIDTH][tc+TILE_WIDTH] = input(j+jj, h+TILE_WIDTH, w+TILE_WIDTH);
            }
            if (tr < kKernel && tc < kKernel) {
              weightShared[jj][tr][tc] = weight(i, j+jj, tr, tc);
            }
          }
          __syncthreads();

          for (int jj = 0; jj < STEP_J; jj++) {
            for (int p = 0; p < kKernel; ++p) {
              for (int q = 0; q < kKernel; ++q) {
                reg += weightShared[jj][p][q] * inputShared[jj][tr + p][tc + q];
              }
            }
          }
          __syncthreads();
        }

        // ReLU
        C[tr][tc] = max(0.f, reg);

        // Max pooling
        __syncthreads();
        if (tr % 2 == 0 && tc % 2 == 0) {
          output(i, h/2, w/2) = max(
              max(C[tr][tc    ], C[tr + 1][tc    ]),
              max(C[tr][tc + 1], C[tr + 1][tc + 1]));
        }
      }
    }
  }
}
