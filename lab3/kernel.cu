#include "hip/hip_runtime.h"
#include "lib/macros.cuh"
#include "kernel.h"

#define TILE_SIZE 32

__global__ void cnn_gpu(float* input,
    float* weight,
    float* bias, float* output)
{
  // Input size: 256 x 224 x 224
  // OUtput size: 256 x 112 x 112
  // GPU specs: 16 SMs, 32 blocks/SM, 2048 threads/SM (32K threads total)

  const int br = blockIdx.y * blockDim.y * 2;
  const int bc = blockIdx.x * blockDim.x * 2;

  const int tr = threadIdx.y;
  const int tc = threadIdx.x;
  const int h = 2*tr;
  const int w = 2*tc;

  __shared__ float inputShared [TILE_SIZE+kKernel-1][TILE_SIZE+kKernel-1] __attribute__((aligned(16 * sizeof(float))));
  __shared__ float weightShared[kKernel            ][kKernel            ] __attribute__((aligned(16 * sizeof(float))));

  for (int i = 0; i < kNum; i++) {
    // Bias
    float C0 = bias[i];
    float C1 = bias[i];
    float C2 = bias[i];
    float C3 = bias[i];

    // Convolution
    for (int j = 0; j < kNum; j++) {
      // Load input and weight submatrices into shared memory
      inputShared[h  ][w  ] = input(j, br+h  , bc+w  );
      inputShared[h  ][w+1] = input(j, br+h  , bc+w+1);
      inputShared[h+1][w  ] = input(j, br+h+1, bc+w  );
      inputShared[h+1][w+1] = input(j, br+h+1, bc+w+1);

      if (tr < kKernel-1) {
        inputShared[TILE_SIZE+tr][w  ] = input(j, br+TILE_SIZE+tr, bc+w  );
        inputShared[TILE_SIZE+tr][w+1] = input(j, br+TILE_SIZE+tr, bc+w+1);
      }
      if (tc < kKernel-1) {
        inputShared[h  ][TILE_SIZE+tc] = input(j, br+h  , bc+TILE_SIZE+tc);
        inputShared[h+1][TILE_SIZE+tc] = input(j, br+h+1, bc+TILE_SIZE+tc);
      }
      if (tr < kKernel-1 && tc < kKernel-1) {
        inputShared[TILE_SIZE+tr][TILE_SIZE+tc] = input(j, br+TILE_SIZE+tr, bc+TILE_SIZE+tc);
      }
      if (tr < kKernel && tc < kKernel) {
        weightShared[tr][tc] = weight(i, j, tr, tc);
      }
      __syncthreads();

      // Matrix multiplication between weight and input submatrices
      for (int p = 0; p < kKernel; p++) {
        for (int q = 0; q < kKernel; q++) {
          C0 += weightShared[p][q] * inputShared[h+p  ][w+q  ];
          C1 += weightShared[p][q] * inputShared[h+p+1][w+q  ];
          C2 += weightShared[p][q] * inputShared[h+p  ][w+q+1];
          C3 += weightShared[p][q] * inputShared[h+p+1][w+q+1];
        }
      }
      __syncthreads();
    }

    // Max pooling + ReLU
    output(i, (br+h)/2, (bc+w)/2) = max(0.f, max(
        max(C0, C1),
        max(C2, C3)));
  }
}
