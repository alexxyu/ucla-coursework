#include "hip/hip_runtime.h"
#include "lib/macros.cuh"
#include "kernel.h"

__global__ void cnn_gpu(float* input,
    float* weight,
    float* bias, float* output)
{
  // your implementation goes here
  // refer to the seq. implementation until you achieve correctness

  // Allocate memory on heap to avoid stack overflow.
  float C[kImSize][kImSize];

  // Input size: 256 x 228 x 228
  // GPU specs: 16 SMs, 32 blocks/SM, 2048 threads/SM (32K threads total)
  // 256 channels -> 256 x 1 x 1 grid (note: we're using 50% of max # of blocks)

  // Bias
  int i = blockIdx.x;
  for (int h = 0; h < kImSize; ++h) {
    for (int w = 0; w < kImSize; ++w)
      C[h][w] = bias[i];
  }

  // Convolution
  for (int j = 0; j < kNum; ++j) {
    for (int h = 0; h < kImSize; ++h) {
      for (int w = 0; w < kImSize; ++w) {
        for (int p = 0; p < kKernel; ++p) {
          for (int q = 0; q < kKernel; ++q)
            C[h][w] += weight(i, j, p, q) * input(j, h + p, w + q);
        }
      }
    }
  }

  // ReLU
  for (int h = 0; h < kImSize; ++h) {
    for (int w = 0; w < kImSize; ++w) {
      C[h][w] = max(0.f, C[h][w]);
    }
  }

  // Max pooling
  for (int h = 0; h < kOutImSize; ++h) {
    for (int w = 0; w < kOutImSize; ++w) {
      output(i, h, w) = max(
          max(C[h * 2][w * 2    ], C[h * 2 + 1][w * 2    ]),
          max(C[h * 2][w * 2 + 1], C[h * 2 + 1][w * 2 + 1]));
    }
  }
}
